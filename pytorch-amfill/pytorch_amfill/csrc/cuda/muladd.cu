#include "hip/hip_runtime.h"
#include <ATen/Operators.h>
#include <torch/all.h>
#include <torch/library.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace pytorch_amfill {

__global__ void amfill_kernel(const int  Xs, const int  Ys,
                              const float* im, const bool* mask, float* om) {

  const int idi = blockIdx.x * blockDim.x + threadIdx.x;
  const int idx = idi % Xs;
  const int idy = idi / Xs;
  if ( idx >= Xs || idy >= Ys )
    return;

  if (mask[idi]) {
    if (im != om)
      om[idi] = im[idi];
    return;
  }

  // find nearest unmasked
  const int mcrad = floor(sqrt(Xs*Xs + Ys*Ys))-1;
  int crad=0; // negative crad indicates first nonmasked pixel
  while ( crad>=0 && crad < mcrad ) {
    crad++;
    const int cr2 = crad*crad;
    for (int dy = -crad ; dy <= crad ; dy++) {
      const int idyy = idy+dy;
      if (idyy<0 || idyy>=Ys)
        continue;
      const int dx = floor( sqrt((float)(cr2-dy*dy)) );
      const int cshft = idyy * Xs + idx;
      if  ( ( idx-dx >= 0   &&  mask[cshft-dx] )
         || ( idx+dx < Xs  &&  mask[cshft+dx] ) ) { // found
        crad *= -1; // negative indicator set
        break;
      }
    }
  }
  if (crad>=0) // may happen only on full mask?
    return;
  const float sigma = -crad ; // final sigma, also negates previous indicator.
  const float sig22 = 2*sigma*sigma;
  const int mrad = floor(2.0*sigma);

  // filter
  float mass=0.0;
  float gsumd=0;
  for (int dy = -mrad ; dy <= mrad ; dy++) {
    const int idyy = idy+dy;
    if (idyy<0 || idyy>=Ys)
      continue;
    const int idiy = idyy * Xs;
    const int dy2 = dy*dy;
    const int xrad = floor(sqrt(2*sig22-dy2));
    for (int dx = -xrad ; dx <= xrad ; dx++) {
      const int idxx = idx+dx;
      const int idii = idiy + idxx;
      if (idxx<0 || idxx>=Xs || ! mask[idii] )
        continue;
      const float wght = exp(-(dx*dx + dy2)/sig22);
      mass += wght;
      gsumd += wght*im[idii];
    }
  }
  om[idi] = mass>0.0 ? gsumd / mass : 0;

}



at::Tensor amfill_cuda(const at::Tensor& im, const at::Tensor& mask) {
  TORCH_CHECK(im.sizes() == mask.sizes());
  TORCH_CHECK(im.dtype() == at::kFloat);
  TORCH_CHECK(mask.dtype() == at::kBool);
  TORCH_INTERNAL_ASSERT(im.device().type() == at::DeviceType::CUDA);
  TORCH_INTERNAL_ASSERT(mask.device().type() == at::DeviceType::CUDA);
  at::Tensor im_contig = im.contiguous();
  at::Tensor mask_contig = mask.contiguous();
  at::Tensor om = at::empty(im_contig.sizes(), im_contig.options());
  const float* im_ptr = im_contig.data_ptr<float>();
  const bool* mask_ptr = mask_contig.data_ptr<bool>();
  float* om_ptr = om.data_ptr<float>();
  int numel = im_contig.numel();
  amfill_kernel<<<(numel+255)/256, 256>>>(im.sizes()[1], im.sizes()[0], im_ptr, mask_ptr, om_ptr);
  return om;
}




at::Tensor amfill_cuda_(at::Tensor& iom, const at::Tensor& mask) {
  TORCH_CHECK(iom.sizes() == mask.sizes());
  TORCH_CHECK(iom.dtype() == at::kFloat);
  TORCH_CHECK(mask.dtype() == at::kBool);
  TORCH_CHECK(iom.is_contiguous());
  TORCH_INTERNAL_ASSERT(iom.device().type() == at::DeviceType::CUDA);
  TORCH_INTERNAL_ASSERT(mask.device().type() == at::DeviceType::CUDA);
  at::Tensor iom_contig = iom.contiguous();
  at::Tensor mask_contig = mask.contiguous();
  float* iom_ptr = iom_contig.data_ptr<float>();
  const bool* mask_ptr = mask_contig.data_ptr<bool>();
  int numel = iom_contig.numel();
  amfill_kernel<<<(numel+255)/256, 256>>>(iom.sizes()[1], iom.sizes()[0], iom_ptr, mask_ptr, iom_ptr);
  return iom;
}


// Registers CUDA implementations
TORCH_LIBRARY_IMPL(pytorch_amfill, CUDA, m) {
  m.impl("amfill", &amfill_cuda);
  m.impl("amfill_", &amfill_cuda_);
}

}

